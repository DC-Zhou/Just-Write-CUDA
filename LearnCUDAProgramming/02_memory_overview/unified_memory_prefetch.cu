#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>


__global__ void add(int n, float *x, float *y)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}
int main()
{
    int N = 1 << 20;
    float *x, *y;
    int device = -1;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipGetDevice(&device);
    // GPU prefetches unified memory memory
    hipMemPrefetchAsync(x, N*sizeof(float), device, nullptr);
    hipMemPrefetchAsync(y, N*sizeof(float), device, nullptr);

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);
    // Host prefetches unified memory memory
    hipMemPrefetchAsync(x, N*sizeof(float), hipCpuDeviceId, nullptr);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for(int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));

    printf("Max error: %f", maxError);

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}