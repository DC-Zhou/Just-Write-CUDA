#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <helper_functions.h>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

void random_init(float *data, int size)
{
    for (int i = 0; i < size; i++)
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param C          matrix C as provided to device
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix C
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////
__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for(int i = 0; i < K; i++)
    {
        sum += A[row * K + i] * B[i * K + col];
    }

    C[row * M + col] = alpha * sum + beta * C[row * M + col];
}

void sgemm_gpu(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);

    sgemm_gpu_kernel <<< dimGrid, dimBlock >>> (A, B, C, N, M, K, alpha, beta);
}

void performance_estimation(void(*sgemm)(const float *, const float *, float *, int, int, int, float, float), const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta) {

    int test_iterations = 100;

    StopWatchInterface *timer = 0;

    sgemm(A, B, C, N, M, K, alpha, beta);

    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for(int i = 0; i < test_iterations; i++) {
        sgemm(A, B, C, N, M, K, alpha, beta);
    }

    sdkStopTimer(&timer);

    float operation_time = sdkGetTimerValue(&timer);
    float operation_per_epoch = operation_time / test_iterations;

    printf("Operation time: %4f ms \n", operation_per_epoch);

    // cleanup
    sdkDeleteTimer(&timer);
}


int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N,M,K;
    float alpha = 2.f;
    float beta  = 1.f;

    N = M = K = 2048;

    A = (float*)malloc(N*K*sizeof(float));
    B = (float*)malloc(K*M*sizeof(float));
    C = (float*)malloc(N*M*sizeof(float));

    // allocation of gpu linear memory
    hipMalloc((void**)&d_A, N*K*sizeof(float));
    hipMalloc((void**)&d_B, K*M*sizeof(float));
    hipMalloc((void**)&d_C, N*M*sizeof(float));

    // initialization of A, B, C
    random_init(A, N*K);
    random_init(B, K*M);
    random_init(C, N*M);

    // copy A, B, C from host to device
    hipMemcpy(d_A, A, N*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K*M*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N*M*sizeof(float), hipMemcpyHostToDevice);

    // do operation
    //    sgemm_gpu(d_A, d_B, d_C, N,M,K,alpha,beta);
    performance_estimation(sgemm_gpu, d_A, d_B, d_C, N,M,K,alpha,beta);

    // copy C from device to host
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}