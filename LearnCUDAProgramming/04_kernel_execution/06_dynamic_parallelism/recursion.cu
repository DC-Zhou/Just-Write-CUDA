#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#define max(a,b) ((a) > (b) ? (a) : (b))

using namespace std;

__global__ void recursive_kernel(int *data, int block_size, int depth) {
    if (depth > 24){
        printf("CUDA does not support more than 24 depth recursion.\n");
        return;
    }

    int x_0 = blockIdx.x * block_size;

    if(depth > 0) {
        __syncthreads();

        if(threadIdx.x == 0) {
            int dimBlock = max(block_size/2, 32);
            int dimGrid  = block_size / dimBlock;

            // prints the calling kernel information
            printf("depth: [%2d], offset: %4d, block_idx: %2d, block_size: %3d\n",
                   depth, x_0, blockIdx.x, block_size);

            recursive_kernel<<< dimGrid, dimBlock>>>(&data[x_0], dimBlock, depth - 1);
            hipDeviceSynchronize();
        }
        __syncthreads();
    }
}

int sum_depth(int depth) {
    if (depth == 1)
        return 1;
    return sum_depth(depth - 1) + depth;
}

int main()
{
    int size = 1 << 12;
    int *data = (int *)malloc(size * sizeof(int));
    int max_depth = 3;

    // create device data
    hipMallocManaged((void **)&data, size * sizeof(int));

    int dimBlock = 512;
    int dimGrid = size / dimBlock;
    recursive_kernel<<<dimGrid, dimBlock>>>(data, dimBlock, max_depth);

    hipDeviceSynchronize();

    // count elements value
    int counter = 0;
    int correct_sum = sum_depth(max_depth);

    for (int i = 0; i < size; i++)
        counter += (data[i] == correct_sum) ? 1 : 0;

    // result
    printf("sum_depth: %d \n", correct_sum);
    if(counter = size)
        printf("Test passed");
    else
        printf("Error!! Obtained %d. It should be %d\n", counter, size);

    hipFree(data);

    return 0;
}