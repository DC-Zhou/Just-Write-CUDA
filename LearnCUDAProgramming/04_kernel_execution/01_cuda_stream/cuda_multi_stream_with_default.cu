
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void foo_kernel(int step)
{
    printf("Loop: %d \n", step);
}

int main()
{
    int n_streams = 5;

    hipStream_t *ls_streams = new hipStream_t[n_streams];

    // create multiple streams
    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&ls_streams[i]);
    }

    // execute kernel in multiple streams each
    for (int i = 0; i < n_streams; i++) {
        if (i == 3)
            foo_kernel<<<1, 1, 0, 0>>>(i);
        else
            foo_kernel<<<1, 1, 0, ls_streams[i]>>>(i);
    }

    // synchronize the host and GPU
    hipDeviceSynchronize();

    // terminates all the created CUDA streams
    for(int i = 0; i < n_streams; i++) {
        hipStreamDestroy(ls_streams[i]);
    }

    return 0;
}