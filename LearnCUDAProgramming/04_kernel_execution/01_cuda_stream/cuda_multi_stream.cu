
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void foo_kernel(int step) {
    printf("Loop: %d \n", step);
}

int main()
{
    int n_streams = 5;
    hipStream_t *ls_streams = new hipStream_t[n_streams];

    // create multiple streams
    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&ls_streams[i]);
    }

    // execute kernel in multiple streams
    for (int i = 0; i < n_streams; i++) {
        foo_kernel<<<1, 1, 0, ls_streams[i]>>>(i);
    }

    // synchronize
    hipDeviceSynchronize();

    // terminates all the created CUDA streams
    for(int i = 0; i < n_streams; i++) {
        hipStreamDestroy(ls_streams[i]);
    }

    delete [] ls_streams;

    return 0;
}