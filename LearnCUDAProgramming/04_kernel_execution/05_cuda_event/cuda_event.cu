#include "hip/hip_runtime.h"
#include <cstdio>
#include <helper_timer.h>

using namespace std;

__global__ void vecadd_kernel(float *c, const float* a, const float *b);
void init_buffer(float *buff, int size);

int main(int argc, char* argv[])
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int buff_size = size * sizeof(float);

    hipHostMalloc((void**)&h_a, buff_size);
    hipHostMalloc((void**)&h_b, buff_size);
    hipHostMalloc((void**)&h_c, buff_size);

    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    hipMalloc((void**)&d_a, buff_size);
    hipMalloc((void**)&d_b, buff_size);
    hipMalloc((void**)&d_c, buff_size);

    // copy host -> device
    hipMemcpy(d_a, h_a, buff_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, buff_size, hipMemcpyHostToDevice);

    // initialize timer
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);

    // initialize cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to measure the execution time
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    // launch cuda kernel
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecadd_kernel<<< dimGrid, dimBlock >>>(d_c, d_a, d_b);

    // record the end time
    hipEventRecord(stop, 0);

    // Synchronize the device
    hipEventSynchronize(stop);
    sdkStopTimer(&timer);

    // copy device -> host
    hipMemcpyAsync(h_c, d_c, buff_size, hipMemcpyDeviceToHost);

    // print estimated kernel execution time
    float elapsed_time_msed = 0.f;
    hipEventElapsedTime(&elapsed_time_msed, start, stop);
    printf("CUDA event estimated - elapsed %.3f ms \n", elapsed_time_msed);

    // Compute and print the performance
    elapsed_time_msed = sdkGetTimerValue(&timer);
    printf("Host measured time= %.3f msec/s\n", elapsed_time_msed);

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    // delete timer
    sdkDeleteTimer(&timer);

    // terminate CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;

}

void init_buffer(float *buff, int size)
{
    for (int i = 0; i < size; i++)
    {
        buff[i] = rand() / (float)RAND_MAX;
    }
}

__global__ void vecadd_kernel(float *c, const float* a, const float *b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}

