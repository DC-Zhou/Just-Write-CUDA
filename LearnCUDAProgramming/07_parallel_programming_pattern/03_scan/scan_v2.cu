#include "hip/hip_runtime.h"
#include "scan.h"

__global__ void scan_v2_kernel(float *d_output, float *d_input, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_data[];
    s_data[threadIdx.x] = d_input[idx];
    s_data[threadIdx.x + BLOCK_DIM] = d_input[idx + BLOCK_DIM];

    int offset = 1;

    while (offset < length) {
        __syncthreads();

        int idx_a = offset * (2 * tid + 1) - 1;
        int idx_b = offset * (2 * tid + 2) - 1;

        if (idx_a >= 0 && idx_b < 2 * BLOCK_DIM) {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d]\t", idx_a, idx_b);
#endif
            s_data[idx_b] += s_data[idx_a];
        }
        offset <<= 1;
#if (DEBUG_INDEX > 0)
        if (tid == 0) printf("\n......................................\n");
#endif
    }

    offset >>= 1;
    while (offset > 0)
    {
        __syncthreads();

        int idx_a = offset * (2 * tid + 2) - 1;
        int idx_b = offset * (2 * tid + 3) - 1;

        if (idx_a >= 0 && idx_b < 2 * BLOCK_DIM)
        {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d]\t", idx_a, idx_b);
#endif
            s_data[idx_b] += s_data[idx_a];
        }

        offset >>= 1;
#if (DEBUG_INDEX > 0)
        if (tid == 0) printf("\n......................................\n");
#endif
    }
    __syncthreads();

    d_output[idx] = s_data[threadIdx.x];
    d_output[idx + BLOCK_DIM] = s_data[threadIdx.x + BLOCK_DIM];
}

void scan_v2(float *d_output, float *d_input, int length)
{
    dim3 dimBlock(BLOCK_DIM);
    dim3 dimGrid((length + BLOCK_DIM - 1) / BLOCK_DIM);

    scan_v2_kernel<<<dimGrid, dimBlock>>>(d_output, d_input, length);
}