#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


__device__ int a[256][512][512];
__device__ float b[256][512][512];

__global__ void grid3D(int nx, int ny, int nz, int id) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= nx || y >= ny || z >= nz) return;

    int array_size = nx * ny * nz;
    int block_size = blockDim.x * blockDim.y * blockDim.z;
    int grid_size = gridDim.x * gridDim.y * gridDim.z;
    int total_threads = block_size * grid_size;

    int thread_rank_in_block = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    int block_rank_in_grid = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
    int thread_rank_in_grid = block_rank_in_grid * block_size + thread_rank_in_block;

    a[z][y][x] = thread_rank_in_grid;
    b[z][y][x] = sqrtf((float) a[z][y][x]);

    if (thread_rank_in_grid == id) {
        printf("array size %3d x %3d x %3d = %d\n",
                 nx,ny,nz, array_size);
        printf("thread block %3d x %3d x %3d = %d\n",
                  blockDim.x, blockDim.y, blockDim.z, block_size);
        printf("thread grid %3d x %3d x %3d = %d\n",
                  gridDim.x, gridDim.y, gridDim.z, grid_size);
        printf("total number of threads in grid %d\n",
                  total_threads);
        printf("a[%d][%d][%d] = %i and b[%d][%d][%d] = %f\n",
                  z, y, x, a[z][y][x], z, y, x, b[z][y][x]);
        printf("for thread with 3D-rank %d 1D-rank %d block rank in grid %d\n", thread_rank_in_grid, thread_rank_in_block,block_rank_in_grid);
    }


}




int main(int argc, char* argv[])
{
    int id = (argc > 1) ? atoi(argv[1]) : 12345;

    dim3 thread3D(32, 8, 2);
    dim3 block3D(16, 64, 128);

    grid3D<<<block3D, thread3D>>>(512,512,256,id);

    return 0;
}

